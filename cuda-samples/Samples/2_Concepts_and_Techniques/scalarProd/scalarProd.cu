#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample calculates scalar products of a
 * given set of input vector pairs
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C" void scalarProdCPU(float *h_C, float *h_A, float *h_B, int vectorN,
                              int elementN);

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

// Total number of input vector pairs; arbitrary
const int VECTOR_N = 4096;
// Number of elements per vector; arbitrary,
// but strongly preferred to be a multiple of warp size
// to meet memory coalescing constraints
const int ELEMENT_N = 16384;
// Total number of data elements
const int DATA_N = VECTOR_N * ELEMENT_N;

const int DATA_SZ = DATA_N * sizeof(float);
const int RESULT_SZ = VECTOR_N * sizeof(float);

///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU, *h_C_GPU_GPT;
    float *d_A, *d_B, *d_C, *d_C_GPT;
    double delta, ref, sum_delta, sum_ref, L1norm, L1norm_GPT;
    StopWatchInterface *hTimer = NULL;
    int i;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest
    // Gflops/s
    findCudaDevice(argc, (const char **)argv);

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    printf("...allocating CPU memory.\n");
    h_A = (float *)malloc(DATA_SZ);
    h_B = (float *)malloc(DATA_SZ);
    h_C_CPU = (float *)malloc(RESULT_SZ);
    h_C_GPU = (float *)malloc(RESULT_SZ);
    h_C_GPU_GPT = (float *)malloc(RESULT_SZ);

    printf("...allocating GPU memory.\n");
    checkCudaErrors(hipMalloc((void **)&d_A, DATA_SZ));
    checkCudaErrors(hipMalloc((void **)&d_B, DATA_SZ));
    checkCudaErrors(hipMalloc((void **)&d_C, RESULT_SZ));
    checkCudaErrors(hipMalloc((void **)&d_C_GPT, RESULT_SZ));

    printf("...generating input data in CPU mem.\n");
    srand(123);

    // Generating input data on CPU
    for (i = 0; i < DATA_N; i++)
    {
        h_A[i] = RandFloat(0.0f, 1.0f);
        h_B[i] = RandFloat(0.0f, 1.0f);
    }

    // Original GPU kernel
    printf("...copying input data to GPU mem.\n");
    // Copy options data to GPU memory for further processing
    checkCudaErrors(hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice));
    printf("Data init done.\n");

    printf("Executing GPU kernel...\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
    scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
    getLastCudaError("scalarProdGPU() execution failed\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    printf("GPU time: %f msecs.\n", sdkGetTimerValue(&hTimer));

    printf("Reading back GPU result...\n");
    // Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost));

    printf("Executing GPU GPT kernel...\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);



    // GPT4scalarProdGPU<<<128, 256>>>(d_C_GPT, d_A, d_B, VECTOR_N, ELEMENT_N);
    // GPT4scalarProdGPUOptimized<<<128, 256>>>(d_C_GPT, d_A, d_B, VECTOR_N, ELEMENT_N);
    // GPT35scalarProdCUDA<<<128, 256>>>(d_C_GPT, d_A, d_B, VECTOR_N, ELEMENT_N);
    // GPT35scalarProdCUDAOptimized<<<128, 256>>>(d_C_GPT, d_A, d_B, VECTOR_N, ELEMENT_N);

    GeminiScalarProdGPU<<<128, 256>>>(d_C_GPT, d_A, d_B, VECTOR_N, ELEMENT_N);
    
    

    getLastCudaError("scalarProdGPU() execution failed\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    printf("GPU GPT time: %f msecs.\n", sdkGetTimerValue(&hTimer));

    printf("Reading back GPU GPT result...\n");
    // Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_C_GPU_GPT, d_C_GPT, RESULT_SZ, hipMemcpyDeviceToHost));

    printf("Checking GPU results...\n");
    printf("..running CPU scalar product calculation\n");
    auto host_start = std::chrono::high_resolution_clock::now();
    scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);
    auto host_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> host_elapsed = host_stop - host_start;
    std::cout << "Host Function Time: " << host_elapsed.count() << " ms" << std::endl;

    printf("...comparing the results for non GPT\n");

    sum_delta = 0;
    sum_ref = 0;

    for (i = 0; i < VECTOR_N; i++)
    {
        delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
        ref = h_C_CPU[i];
        sum_delta += delta;
        sum_ref += ref;
    }

    L1norm = sum_delta / sum_ref;

    printf("...comparing the results for GPT\n");
    sum_delta = 0;
    sum_ref = 0;

    for (i = 0; i < VECTOR_N; i++)
    {
        delta = fabs(h_C_GPU_GPT[i] - h_C_CPU[i]);
        ref = h_C_CPU[i];
        sum_delta += delta;
        sum_ref += ref;
    }

    L1norm_GPT = sum_delta / sum_ref;

    printf("Shutting down...\n");
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_A));
    free(h_C_GPU);
    free(h_C_GPU_GPT);
    free(h_C_CPU);
    free(h_B);
    free(h_A);
    sdkDeleteTimer(&hTimer);

    printf("L1 error: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "Test passed\n" : "Test failed!\n");
    printf("L1 GPT error: %E\n", L1norm_GPT);
    printf((L1norm_GPT < 1e-6) ? "Test passed\n" : "Test failed!\n");
}
