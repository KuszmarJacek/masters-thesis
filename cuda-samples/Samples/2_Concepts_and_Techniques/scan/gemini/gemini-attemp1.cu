#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>

#define THREADBLOCK_SIZE 256
#define BLOCK_SIZE 256

__global__ void scanExclusiveKernel(uint *dst, uint *src, uint batchSize, uint arrayLength) {
    // Thread identifiers
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y;

    // Shared memory for partial scan within a block
    __shared__ uint sharedScan[BLOCK_SIZE];

    // Handle potential out-of-bounds threads
    if (idx < arrayLength) {
        // Load the current element from the input array
        sharedScan[idx] = src[idx * batchSize + idy];
    } else {
        // Out-of-bounds threads contribute zero to the scan
        sharedScan[idx] = 0;
    }

    // Perform parallel prefix sum (exclusive scan) within each block
    for (int d = 1; d < BLOCK_SIZE; d *= 2) {
        if (idx < arrayLength - d) {
            sharedScan[idx] += sharedScan[idx + d];
        }
    }

    // Handle potential odd stages and write the final result
    for (int d = BLOCK_SIZE / 2; d > 0; d /= 2) {
        if (idx >= d && idx < arrayLength - d) {
            sharedScan[idx] = sharedScan[idx - d] + sharedScan[idx];
        }
    }

    // Write the exclusive scan results back to the destination array
    if (idx < arrayLength) {
        dst[(idx * batchSize + idy)] = sharedScan[idx];
    }
}


// Sequential version of the exclusive scan for verification
void scanExclusiveHost(uint *dst, uint *src, uint batchSize, uint arrayLength) {
    for (uint i = 0; i < batchSize; i++, src += arrayLength, dst += arrayLength) {
        dst[0] = 0;
        for (uint j = 1; j < arrayLength; j++) {
            dst[j] = src[j - 1] + dst[j - 1];
        }
    }
}

int main() {
    // Number of arrays and length of each array
    uint batchSize = 256;
    uint threadsPerBlock = 256;
    // uint arrayLength = 607;
    // uint arrayLength = 16384;
    // uint arrayLength = 16384 * 2;
    // uint arrayLength = 4096;
    uint arrayLength = 512;

    // Allocate memory for host arrays
    uint *h_src = new uint[batchSize * arrayLength];
    uint *h_dst_gpu = new uint[batchSize * arrayLength];
    uint *h_dst_host = new uint[batchSize * arrayLength];

    // Initialize source data
    for (uint i = 0; i < batchSize * arrayLength; ++i) {
        h_src[i] = rand() % 10;  // Random numbers for example
    }

    // Allocate memory for device arrays
    uint *d_src, *d_dst;
    hipMalloc(&d_src, batchSize * arrayLength * sizeof(uint));
    hipMalloc(&d_dst, batchSize * arrayLength * sizeof(uint));

    // Copy source data from host to device
    hipMemcpy(d_src, h_src, batchSize * arrayLength * sizeof(uint), hipMemcpyHostToDevice);

    // Declare CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording the GPU time
    hipEventRecord(start);

    // scanExclusiveKernel1<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
    //                     THREADBLOCK_SIZE>>>(d_dst, d_src, 4 * THREADBLOCK_SIZE);

    // scanExclusiveKernel<<<(batchSize + threadsPerBlock - 1) / threadsPerBlock, arrayLength>>>(d_dst, d_src, batchSize, arrayLength);

    scanExclusiveKernel<<<batchSize, arrayLength, 1024 * sizeof(uint)>>>(d_dst, d_src, batchSize, arrayLength);

    // ----------------------------------------------------------------------------------------------------------------------


    // scanExclusiveKernel<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
    //                     THREADBLOCK_SIZE>>>(d_dst, d_src, batchSize, arrayLength);

    // dim3 gridSize(1024, 1024); // Adjust based on your data size and GPU architecture
    // dim3 blockSize(256);   // Adjust based on shared memory size and data type

    // // Launch the kernel
    // scanExclusiveKernel<<<gridSize, blockSize>>>(d_dst, d_src, batchSize, arrayLength);

    hipDeviceSynchronize();

    // Stop recording the GPU time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Kernel Time: " << milliseconds << " ms" << std::endl;

    // Check for kernel errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    // Copy results back to host
    hipMemcpy(h_dst_gpu, d_dst, batchSize * arrayLength * sizeof(uint), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // Measure host execution time
    auto host_start = std::chrono::high_resolution_clock::now();
    
    // Perform the sequential scan on the host
    scanExclusiveHost(h_dst_host, h_src, batchSize, arrayLength);

    auto host_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> host_elapsed = host_stop - host_start;
    std::cout << "Host Function Time: " << host_elapsed.count() << " ms" << std::endl;

    // Compare results
    bool correct = true;
    for (uint i = 0; i < batchSize * arrayLength; ++i) {
        if (h_dst_gpu[i] != h_dst_host[i]) {
            std::cerr << "Mismatch at index " << i << ": GPU " << h_dst_gpu[i] << ", Host " << h_dst_host[i] << std::endl;
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "Test Passed: GPU results match the host results." << std::endl;
    } else {
        std::cout << "Test Failed: GPU results do not match the host results." << std::endl;
    }

    // Free device memory
    hipFree(d_src);
    hipFree(d_dst);

    // Free host memory
    delete[] h_src;
    delete[] h_dst_gpu;
    delete[] h_dst_host;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

