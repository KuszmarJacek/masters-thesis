#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>


__global__ void scanExclusiveKernel(uint *dst, uint *src, uint arrayLength) {
  // Get the index of the current array based on block index
  uint i = blockIdx.x;

  // Compute the starting index for src and dst arrays for this block
  uint startIdx = i * arrayLength;
  src += startIdx;
  dst += startIdx;

  // Thread index within the block represents the element index within the array
  uint j = threadIdx.x;

  // Shared memory for inter-thread communication within the block
  extern __shared__ uint temp[];

  // Load input into shared memory for faster access
  if (j < arrayLength) {
    temp[j] = src[j];
  }
  __syncthreads(); // Ensure all data is loaded into shared memory

  // Initialize the first element of the block's output to 0
  if (j == 0) {
    dst[0] = 0;
  }

  // Perform scan using shared memory
  if (j < arrayLength) {
    uint sum = 0;
    for (int k = 1; k <= j; k++) {
      sum += temp[k - 1];
    }
    dst[j] = sum;
  }
}

// Sequential version of the exclusive scan for verification
void scanExclusiveHost(uint *dst, uint *src, uint batchSize, uint arrayLength) {
    for (uint i = 0; i < batchSize; i++, src += arrayLength, dst += arrayLength) {
        dst[0] = 0;
        for (uint j = 1; j < arrayLength; j++) {
            dst[j] = src[j - 1] + dst[j - 1];
        }
    }
}


#define THREADBLOCK_SIZE 256

int main() {
    // Number of arrays and length of each array
    uint batchSize = 256;
    uint arrayLength = 1024;

    // Allocate memory for host arrays
    uint *h_src = new uint[batchSize * arrayLength];
    uint *h_dst_gpu = new uint[batchSize * arrayLength];
    uint *h_dst_host = new uint[batchSize * arrayLength];

    // Initialize source data
    for (uint i = 0; i < batchSize * arrayLength; ++i) {
        h_src[i] = rand() % 10;  // Random numbers for example
    }

    // Allocate memory for device arrays
    uint *d_src, *d_dst;
    hipMalloc(&d_src, batchSize * arrayLength * sizeof(uint));
    hipMalloc(&d_dst, batchSize * arrayLength * sizeof(uint));

    // Copy source data from host to device
    hipMemcpy(d_src, h_src, batchSize * arrayLength * sizeof(uint), hipMemcpyHostToDevice);

    // Declare CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording the GPU time
    hipEventRecord(start);

    // Launch the kernel
    scanExclusiveKernel<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
                        THREADBLOCK_SIZE>>>(d_dst, d_src, arrayLength);
    hipDeviceSynchronize();

    // Stop recording the GPU time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU Kernel Time: " << milliseconds << " ms" << std::endl;

    // Check for kernel errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    // Copy results back to host
    hipMemcpy(h_dst_gpu, d_dst, batchSize * arrayLength * sizeof(uint), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // Measure host execution time
    auto host_start = std::chrono::high_resolution_clock::now();
    
    // Perform the sequential scan on the host
    scanExclusiveHost(h_dst_host, h_src, batchSize, arrayLength);

    auto host_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> host_elapsed = host_stop - host_start;
    std::cout << "Host Function Time: " << host_elapsed.count() << " ms" << std::endl;

    // Compare results
    bool correct = true;
    for (uint i = 0; i < batchSize * arrayLength; ++i) {
        if (h_dst_gpu[i] != h_dst_host[i]) {
            std::cerr << "Mismatch at index " << i << ": GPU " << h_dst_gpu[i] << ", Host " << h_dst_host[i] << std::endl;
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "Test Passed: GPU results match the host results." << std::endl;
    } else {
        std::cout << "Test Failed: GPU results do not match the host results." << std::endl;
    }

    // Free device memory
    hipFree(d_src);
    hipFree(d_dst);

    // Free host memory
    delete[] h_src;
    delete[] h_dst_gpu;
    delete[] h_dst_host;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
