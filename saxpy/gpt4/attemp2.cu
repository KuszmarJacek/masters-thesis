#include "hip/hip_runtime.h"
// Rewrite this kernel using grid stride loop?



#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy_kernel_grid_stride(int n, float a, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Total number of threads in the grid
    for (int i = index; i < n; i += stride)
        y[i] = a * x[i] + y[i];
}

void saxpy(int n, float a, float *x, float *y)
{
    float *d_x, *d_y;

    // Allocate GPU memory
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Execution configuration
    int blockSize = 256;  // Number of threads per block
    // A heuristic to use approximately the number of SMs on the device
    int numBlocks = min((n + blockSize - 1) / blockSize, 65535);

    // Launch the kernel with a grid-stride loop
    saxpy_kernel_grid_stride<<<numBlocks, blockSize>>>(n, a, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}

int main()
{
    int n = 1 << 20;  // Example size
    float a = 2.0f;
    float *x, *y;

    // Allocate memory on the host
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(n * sizeof(float));

    // Initialize data
    for(int i = 0; i < n; i++) {
        x[i] = 1.0f;  // Example values
        y[i] = 2.0f;
    }

    // Call SAXPY function
    saxpy(n, a, x, y);

    // Free host memory
    free(x);
    free(y);

    return 0;
}
