// Replace the kernel with an already implemented saxpy from cublas library.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void saxpy(int n, float a, float *x, float *y)
{
    float *d_x, *d_y;
    hipblasHandle_t handle;

    // Initialize CUBLAS context
    hipblasCreate(&handle);

    // Allocate GPU memory
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Call cublasSaxpy
    // cublasSaxpy(handle, n, &a, d_x, 1, d_y, 1);
    hipblasSaxpy(handle, n, &a, d_x, 1, d_y, 1);

    // Copy result back to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);
}

int main()
{
    int n = 1 << 20;  // Example size
    float a = 2.0f;
    float *x, *y;

    // Allocate memory on the host
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(n * sizeof(float));

    // Initialize data
    for(int i = 0; i < n; i++) {
        x[i] = 1.0f;  // Example values
        y[i] = 2.0f;
    }

    // Call SAXPY function using cuBLAS
    saxpy(n, a, x, y);

    // Free host memory
    free(x);
    free(y);

    return 0;
}
