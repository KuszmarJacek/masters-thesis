// Rewrite provided sequential code in C to a CUDA kernel. Assume that n = 1 << 20;
// void saxpy(int n, float a, float *x, float *y)
// {
//     for (int i = 0; i < n; ++i)
//         y[i] = a * x[i] + y[i];
// }

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy_kernel(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

void saxpy(int n, float a, float *x, float *y)
{
    float *d_x, *d_y;

    // Allocate GPU memory
    hipMalloc((void **)&d_x, n * sizeof(float));
    hipMalloc((void **)&d_y, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Execution configuration
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (n + blockSize - 1) / blockSize;  // Number of blocks

    // Launch the kernel
    saxpy_kernel<<<numBlocks, blockSize>>>(n, a, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}

int main()
{
    int n = 1 << 20;  // Example size
    float a = 2.0f;
    float *x, *y;

    // Allocate memory on the host
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(n * sizeof(float));

    // Initialize data
    for(int i = 0; i < n; i++) {
        x[i] = 1.0f;  // Example values
        y[i] = 2.0f;
    }

    // Call SAXPY function
    saxpy(n, a, x, y);

    // Free host memory
    free(x);
    free(y);

    return 0;
}
