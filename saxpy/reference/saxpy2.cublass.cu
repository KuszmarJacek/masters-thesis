
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

int main(void)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int n = 1 << 20;
    float *x, *y, alpha = 2.0;
    hipMalloc(&x, n * sizeof(float));
    hipMalloc(&y, n * sizeof(float));
    hipblasSaxpy(handle, n, &alpha, x, 1, y, 1);
    hipblasDestroy(handle);
    hipFree(x);
    hipFree(y);
}