#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <shared.h>

#define BLOCK_SIZE 16

/*
 * prints matrices
 * Because matrices filled with dummy 0s function takes 3 dim arguments:
 *      actual x and y dimension and dim as big square matrix's dimension
 */
void print_matrices(float* matrix, char* file_Name, int x_dim, int y_dim, int dim)
{
    std::ofstream outFile;
    outFile.open (file_Name);

    outFile << std::fixed;
    outFile << std::setprecision(2);

    for (int i = 0; i < x_dim; i++) {

        for (int j = 0; j < y_dim; j++) {
            outFile << matrix[i * dim + j] << " ";
        }
        outFile << std::endl;
    }
}

//naive CPU matrix multiplication code
//because of its simplicity directly taken from web
//it multiplies square matrices
__host__ void cpu_matrix_mult(float *h_a, float *h_b, float *h_result, int m) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < m; ++j)
        {
            float tmp = 0.0;
            for (int h = 0; h < m; ++h)
            {
                tmp += h_a[i * m + h] * h_b[h * m + j];
            }
            h_result[i * m + j] = tmp;
        }
    }
}

//this function is for filling the matrices with cos and sin values randomly
//I transform the matrices to square matrix in order to perform better multiplication
__host__ int fill(float **Lmatrix, float **Rmatrix, int LdimX, int LdimY, int RdimX, int RdimY) {

    int sqr_dim_X, sqr_dim_Y, size;

    sqr_dim_X = RdimX;
    if (LdimX > RdimX) {
        sqr_dim_X = LdimX;
    }

    sqr_dim_Y = RdimY;
    if (LdimY > RdimY) {
        sqr_dim_Y = LdimY;
    }

    size = sqr_dim_Y;
    if (sqr_dim_X > sqr_dim_Y) {
        size = sqr_dim_X;
    }

    int temp = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1);
    size = temp * BLOCK_SIZE;

    size_t pt_size = size * size * sizeof(float);

    *Lmatrix = (float *) malloc(pt_size);
    *Rmatrix = (float *) malloc(pt_size);

    memset(*Lmatrix, 0, pt_size);
    memset(*Rmatrix, 0, pt_size);

    for (int i = 0; i < LdimX; i++) {
        for (int j = 0; j < LdimY; j++) {
            int dummy = size * i + j;
            (*Lmatrix)[dummy] = sinf(dummy);
        }
    }
    for (int i = 0; i < RdimX; i++) {
        for (int j = 0; j < RdimY; j++) {
            int dummy = size * i + j;
            (*Rmatrix)[dummy] = cosf(dummy);
        }
    }
    return size;
}

__global__ void Gemini_matrix_mult(float *h_a, float *h_b, float *h_result, int m) {
  // Thread and block indices
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Shared memory tile
  __shared__ float s_a[16][16];
  __shared__ float s_b[16][16];

  // Block size (assuming square block)
  int block_size = blockDim.x;

  // Calculate tile coordinates within the global matrix
  int i = by * block_size + ty;
  int j = bx * block_size + tx;

  // Loop through tiles
  for (int a_i = 0; a_i < m; a_i += block_size) {
    for (int a_j = 0; a_j < m; a_j += block_size) {
      // Load tile elements from global memory to shared memory (coalesced access)
      if (i < m && a_i + ty < m) {
        s_a[ty][tx] = h_a[(i + ty) * m + a_i + tx];
      }
      if (j < m && a_j + tx < m) {
        s_b[ty][tx] = h_b[(a_j + tx) * m + j + ty];
      }
      // Synchronize threads to ensure all data is loaded before computation
      __syncthreads();

      // Perform dot product within the tile
      float tmp = 0.0f;
      for (int k = 0; k < block_size; ++k) {
        tmp += s_a[ty][k] * s_b[k][tx];
      }

      // Synchronize threads again before writing to global memory
      __syncthreads();

      // Write the result to global memory (avoid race conditions)
      if (i < m && j < m) {
        h_result[i * m + j] = tmp;
      }
    }
  }
}

int main(void)
{
    //size of the vectors to be processed  and matrix dimensions
    int Left_matrix_x  = 512; 
    int Left_matrix_y  = 512; 
    int Right_matrix_x = 512; 
    int Right_matrix_y = 512;

    // int Left_matrix_x  = 1024; 
    // int Left_matrix_y  = 1024; 
    // int Right_matrix_x = 1024; 
    // int Right_matrix_y = 1024; 

    // int Left_matrix_x  = 1024 * 2; 
    // int Left_matrix_y  = 1024 * 2; 
    // int Right_matrix_x = 1024 * 2; 
    // int Right_matrix_y = 1024 * 2; 

    // int Left_matrix_x  = 1024 * 4; 
    // int Left_matrix_y  = 1024 * 4; 
    // int Right_matrix_x = 1024 * 4; 
    // int Right_matrix_y = 1024 * 4; 


    int Left_vector_size;
    int Right_vector_size;

    float *Left_Vector_h, *Right_Vector_h, *Left_Vector_d, *Right_Vector_d, *Res_h, *Res_d, *CPU;  // Pointer to host & device arrays

    int dim = fill(&Left_Vector_h, &Right_Vector_h, Left_matrix_x, Left_matrix_y, Right_matrix_x, Right_matrix_y); //fills the matrices with random values

    print_matrices(Left_Vector_h,"Input_LHS",Left_matrix_x,Left_matrix_y,dim);
    print_matrices(Right_Vector_h,"Input_RHS",Right_matrix_x,Right_matrix_y,dim);

    size_t vector_size;
    vector_size = dim*dim * sizeof(float);

    Res_h = (float *) malloc(vector_size); // Allocate array on host for result
    CPU = (float *) malloc(vector_size);// Allocate array on host for CPU_matrix_multiplication result

    hipMalloc((void **) &Left_Vector_d, vector_size);     // Allocate array on device for LHS operand
    hipMalloc((void **) &Right_Vector_d, vector_size);   // Allocate array on device for RHS operand but this is vector 1xN
    hipMalloc((void **) &Res_d, vector_size);     // Allocate array on device for result

    hipMemcpy(Left_Vector_d, Left_Vector_h, vector_size, hipMemcpyHostToDevice);      // copy values to device
    hipMemcpy(Right_Vector_d, Right_Vector_h, vector_size, hipMemcpyHostToDevice);   // copy values to device

    //Block dimension is directly from block_size
    dim3 Block_dim(BLOCK_SIZE, BLOCK_SIZE);
    //Grid dimension is found by dividing matrix dimension to block_size
    dim3 Grid_dim(dim / BLOCK_SIZE, dim / BLOCK_SIZE);

    //commented out the functions which helps to calculate time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //kernel call
    Gemini_matrix_mult<< < Grid_dim, Block_dim >> > (Left_Vector_d, Right_Vector_d, Res_d, dim);

    //commented out the functions which helps to calculate time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Retrieve result from device and store it in host array
    hipMemcpy(Res_h, Res_d, vector_size, hipMemcpyDeviceToHost);

    clock_t begin = clock();

    cpu_matrix_mult(Left_Vector_h,Right_Vector_h,CPU,dim); //matrix multiplication on cpu

    clock_t end = clock();
    double time_spent = (double)1000*(end - begin) / CLOCKS_PER_SEC;

    //commented out the functions which helps to calculate time
    printf("GPU time= %f ms\n", et);

    printf("CPU time= %lf ms\n", time_spent);

    //Prints the results
    print_matrices(Res_h,"GPU_out",Left_matrix_x,Right_matrix_y,dim);
    print_matrices(CPU,"CPU_out",Left_matrix_x,Right_matrix_y,dim);

    bool eqaul = true;
    for (int i=0;i< Left_matrix_x && eqaul;i++){
        for (int j = 0; j < Right_matrix_y && eqaul; j++) {
            if (abs(Res_h[i*dim+j]-CPU[i*dim+j]) > 0.001)
            {
                eqaul = false;
                printf("NOT EQUAL\n");
            }
        }
    }
    if (eqaul)
    {
        std::cout<<"Results are equal!"<<std::endl;
    }
    else
    {
        std::cout<<"Results are NOT equal!"<<std::endl;
    }

    // Cleanup
    free(Left_Vector_h);
    free(Right_Vector_h);
    free(Res_h);
    free(CPU);
    hipFree(Left_Vector_d);
    hipFree(Right_Vector_d);
    hipFree(Res_d);
}