#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#define BLOCK_SIZE 16
#define TILE_WIDTH 16

/*
 * prints matrices
 * Because matrices filled with dummy 0s function takes 3 dim arguments:
 *      actual x and y dimension and dim as big square matrix's dimension
 */
void print_matrices(float* matrix, char* file_Name, int x_dim, int y_dim, int dim)
{
    std::ofstream outFile;
    outFile.open (file_Name);

    outFile << std::fixed;
    outFile << std::setprecision(2);

    for (int i = 0; i < x_dim; i++) {

        for (int j = 0; j < y_dim; j++) {
            outFile << matrix[i * dim + j] << " ";
        }
        outFile << std::endl;
    }
}

//naive CPU matrix multiplication code
//because of its simplicity directly taken from web
//it multiplies square matrices
__host__ void cpu_matrix_mult(float *h_a, float *h_b, float *h_result, int m) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < m; ++j)
        {
            float tmp = 0.0;
            for (int h = 0; h < m; ++h)
            {
                tmp += h_a[i * m + h] * h_b[h * m + j];
            }
            h_result[i * m + j] = tmp;
        }
    }
}

//this function is for filling the matrices with cos and sin values randomly
//I transform the matrices to square matrix in order to perform better multiplication
__host__ int fill(float **Lmatrix, float **Rmatrix, int LdimX, int LdimY, int RdimX, int RdimY) {

    int sqr_dim_X, sqr_dim_Y, size;

    sqr_dim_X = RdimX;
    if (LdimX > RdimX) {
        sqr_dim_X = LdimX;
    }

    sqr_dim_Y = RdimY;
    if (LdimY > RdimY) {
        sqr_dim_Y = LdimY;
    }

    size = sqr_dim_Y;
    if (sqr_dim_X > sqr_dim_Y) {
        size = sqr_dim_X;
    }

    int temp = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1);
    size = temp * BLOCK_SIZE;

    size_t pt_size = size * size * sizeof(float);

    *Lmatrix = (float *) malloc(pt_size);
    *Rmatrix = (float *) malloc(pt_size);

    memset(*Lmatrix, 0, pt_size);
    memset(*Rmatrix, 0, pt_size);

    for (int i = 0; i < LdimX; i++) {
        for (int j = 0; j < LdimY; j++) {
            int dummy = size * i + j;
            (*Lmatrix)[dummy] = sinf(dummy);
        }
    }
    for (int i = 0; i < RdimX; i++) {
        for (int j = 0; j < RdimY; j++) {
            int dummy = size * i + j;
            (*Rmatrix)[dummy] = cosf(dummy);
        }
    }
    return size;
}

__global__ void GPT4matrixMultTiled(float *d_a, float *d_b, float *d_result, int m) {
    // Allocate 2D tiles in shared memory
    __shared__ float tile_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tile_b[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float tmp = 0.0;

    // Loop over the tiles of the input matrices
    for (int k = 0; k < (m - 1) / TILE_WIDTH + 1; ++k) {
        // Load the tiles into shared memory
        if (row < m && (k*TILE_WIDTH + threadIdx.x) < m)
            tile_a[threadIdx.y][threadIdx.x] = d_a[row * m + k * TILE_WIDTH + threadIdx.x];
        else
            tile_a[threadIdx.y][threadIdx.x] = 0.0;

        if (col < m && (k*TILE_WIDTH + threadIdx.y) < m)
            tile_b[threadIdx.y][threadIdx.x] = d_b[(k * TILE_WIDTH + threadIdx.y) * m + col];
        else
            tile_b[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Perform the multiplication for the tile
        for (int n = 0; n < TILE_WIDTH; ++n) {
            tmp += tile_a[threadIdx.y][n] * tile_b[n][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the result to global memory
    if (row < m && col < m)
        d_result[row * m + col] = tmp;
}

// main routine that executes on the host
int main(void)
{
    //size of the vectors to be processed  and matrix dimensions
    int Left_matrix_x  = 512; 
    int Left_matrix_y  = 512; 
    int Right_matrix_x = 512; 
    int Right_matrix_y = 512;

    // int Left_matrix_x  = 1024; 
    // int Left_matrix_y  = 1024; 
    // int Right_matrix_x = 1024; 
    // int Right_matrix_y = 1024; 

    // int Left_matrix_x  = 1024 * 2; 
    // int Left_matrix_y  = 1024 * 2; 
    // int Right_matrix_x = 1024 * 2; 
    // int Right_matrix_y = 1024 * 2; 

    // int Left_matrix_x  = 1024 * 4; 
    // int Left_matrix_y  = 1024 * 4; 
    // int Right_matrix_x = 1024 * 4; 
    // int Right_matrix_y = 1024 * 4; 


    int Left_vector_size;
    int Right_vector_size;

    float *Left_Vector_h, *Right_Vector_h, *Left_Vector_d, *Right_Vector_d, *Res_h, *Res_d, *CPU;  // Pointer to host & device arrays

    // printf("Enter m n n k :\n");

    // scanf("%d %d %d %d",&Left_matrix_x,&Left_matrix_y,&Right_matrix_x,&Right_matrix_y); // input matrix dimensions are taken

    int dim = fill(&Left_Vector_h, &Right_Vector_h, Left_matrix_x, Left_matrix_y, Right_matrix_x, Right_matrix_y); //fills the matrices with random values

    print_matrices(Left_Vector_h,"Input_LHS",Left_matrix_x,Left_matrix_y,dim);
    print_matrices(Right_Vector_h,"Input_RHS",Right_matrix_x,Right_matrix_y,dim);

    size_t vector_size;
    vector_size = dim*dim * sizeof(float);

    Res_h = (float *) malloc(vector_size); // Allocate array on host for result
    CPU = (float *) malloc(vector_size);// Allocate array on host for CPU_matrix_multiplication result

    hipMalloc((void **) &Left_Vector_d, vector_size);     // Allocate array on device for LHS operand
    hipMalloc((void **) &Right_Vector_d, vector_size);   // Allocate array on device for RHS operand but this is vector 1xN
    hipMalloc((void **) &Res_d, vector_size);     // Allocate array on device for result

    hipMemcpy(Left_Vector_d, Left_Vector_h, vector_size, hipMemcpyHostToDevice);      // copy values to device
    hipMemcpy(Right_Vector_d, Right_Vector_h, vector_size, hipMemcpyHostToDevice);   // copy values to device

    //Block dimension is directly from block_size
    dim3 Block_dim(BLOCK_SIZE, BLOCK_SIZE);
    //Grid dimension is found by dividing matrix dimension to block_size
    dim3 Grid_dim(dim / BLOCK_SIZE, dim / BLOCK_SIZE);

    //commented out the functions which helps to calculate time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //kernel call
    GPT4matrixMultTiled<< < Grid_dim, Block_dim >> > (Left_Vector_d, Right_Vector_d, Res_d, dim);

    //commented out the functions which helps to calculate time
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Retrieve result from device and store it in host array
    hipMemcpy(Res_h, Res_d, vector_size, hipMemcpyDeviceToHost);

    clock_t begin = clock();

    cpu_matrix_mult(Left_Vector_h,Right_Vector_h,CPU,dim); //matrix multiplication on cpu

    clock_t end = clock();
    double time_spent = (double)1000*(end - begin) / CLOCKS_PER_SEC;

    //commented out the functions which helps to calculate time
    printf("GPU time= %f ms\n", et);

    printf("CPU time= %lf ms\n", time_spent);

    //Prints the results
    print_matrices(Res_h,"GPU_out",Left_matrix_x,Right_matrix_y,dim);
    print_matrices(CPU,"CPU_out",Left_matrix_x,Right_matrix_y,dim);

    bool eqaul = true;
    for (int i=0;i< Left_matrix_x && eqaul;i++){
        for (int j = 0; j < Right_matrix_y && eqaul; j++) {
            if (abs(Res_h[i*dim+j]-CPU[i*dim+j]) > 0.001)
            {
                eqaul = false;
                printf("NOT EQUAL\n");
            }
        }
    }
    if (eqaul)
    {
        std::cout<<"Results are equal!"<<std::endl;
    }
    else
    {
        std::cout<<"Results are NOT equal!"<<std::endl;
    }

    // Cleanup
    free(Left_Vector_h);
    free(Right_Vector_h);
    free(Res_h);
    free(CPU);
    hipFree(Left_Vector_d);
    hipFree(Right_Vector_d);
    hipFree(Res_d);
}